#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

__device__ void d_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != '\0') {
        i++;
    }

    int j = 0;
    while (src[j] != '\0') {
        dest[i++] = src[j++];
    }

    dest[i] = '\0';
}


/**
 * @brief Function to search for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
 *@param prev_block_hash The hash of the previous block
*/
__global__ void findNonce(const BYTE *prev_block_hash, const BYTE *top_hash, const BYTE *difficulty, uint64_t max_nonce, BYTE *best_hash, uint64_t *best_nonce, bool *found) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (idx > max_nonce || *found) {
        return;
    }

    BYTE hash[SHA256_HASH_SIZE];
    BYTE block_content[BLOCK_SIZE];
    char nonce_str[20];

    d_strcpy((char*)block_content, (const char*)prev_block_hash);
    d_strcat((char*)block_content, (const char*)top_hash);

    intToString(idx, nonce_str);

    d_strcat((char*)block_content, nonce_str);
    
    BYTE local_difficulty[SHA256_HASH_SIZE];
    memcpy(local_difficulty, difficulty, SHA256_HASH_SIZE);


    apply_sha256(block_content, d_strlen((const char*)block_content), hash, 1);

     if (compare_hashes(hash, local_difficulty) <= 0) {
        memcpy(best_hash, hash, SHA256_HASH_SIZE);
        *best_nonce = idx;
        *found = true;
    }
}


int main(int argc, char **argv) {
    BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE];
    uint64_t nonce = 0, *d_best_nonce;
    BYTE *d_prev_block_hash, *d_top_hash, *d_difficulty, *d_best_hash;
    bool *d_found, found = false;
    hipMalloc(&d_found, sizeof(bool));
    hipMemcpy(d_found, &found, sizeof(bool), hipMemcpyHostToDevice);

    apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
    apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
    apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
    apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);

    strcpy((char *)tx12, (const char *)hashed_tx1);
    strcat((char *)tx12, (const char *)hashed_tx2);
    apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
    strcpy((char *)tx34, (const char *)hashed_tx3);
    strcat((char *)tx34, (const char *)hashed_tx4);
    apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
    strcpy((char *)tx1234, (const char *)hashed_tx12);
    strcat((char *)tx1234, (const char *)hashed_tx34);
    apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

    hipMalloc(&d_prev_block_hash, SHA256_HASH_SIZE);
    hipMalloc(&d_top_hash, SHA256_HASH_SIZE);
    hipMalloc(&d_difficulty, SHA256_HASH_SIZE);
    hipMalloc(&d_best_hash, SHA256_HASH_SIZE);
    hipMalloc(&d_best_nonce, sizeof(uint64_t));

    hipMemcpy(d_prev_block_hash, prev_block_hash, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_top_hash, top_hash, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_difficulty, difficulty_5_zeros, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_best_hash, difficulty_5_zeros, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_best_nonce, &nonce, sizeof(uint64_t), hipMemcpyHostToDevice);

    int numBlocks = MAX_NONCE / 256;
    hipEvent_t start, stop;
	startTiming(&start, &stop);

    findNonce<<<numBlocks, 256>>>(d_prev_block_hash, d_top_hash, d_difficulty, MAX_NONCE, d_best_hash, d_best_nonce, d_found);
    hipDeviceSynchronize();
    float seconds = stopTiming(&start, &stop);

    hipMemcpy(&nonce, d_best_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);
    BYTE block_hash[SHA256_HASH_SIZE];
    hipMemcpy(block_hash, d_best_hash, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);

    printResult(block_hash, nonce, seconds);
    hipFree(d_prev_block_hash);
    hipFree(d_top_hash);
    hipFree(d_difficulty);
    hipFree(d_best_hash);
    hipFree(d_best_nonce);
    hipFree(d_found);

    return 0;
}
